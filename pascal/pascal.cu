#include "hip/hip_runtime.h"
#include "miner.h"
#include "cuda_helper.h"

void pascal_cpu_init(int thr_id);
void pascal_cpu_hash(int thr_id, uint32_t threads, uint32_t startnonce, uint32_t nonceoffset, uint32_t *ms, uint32_t *const result);
void pascal_midstate(const uint32_t *data, uint32_t *midstate);
void copydata(const uint32_t *data);

#define rrot(x, n)	ROTR32(x, n)

bool fulltest_pascal(const uint32_t *vhash64, const uint32_t *ptarget)
{
	for(int i = 0; i < 8; i++)
	{
		if(vhash64[i] > ptarget[7 - i])
			return false;
		else
			if(vhash64[i] < ptarget[7 - i])
				return true;
	}
	return true;
}

void pascal_hash(uint32_t *output, const uint32_t *data, uint32_t datasize, uint32_t nonce, const uint32_t *midstate)
{
	int i;
	uint32_t s0, s1, t1, t2, maj, ch, a, b, c, d, e, f, g, h;
	uint32_t w[64];

	const uint32_t k[64] = {
		0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
		0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
		0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
		0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
		0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
		0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
		0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
		0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
	};
	const uint32_t hc[8] = {
		0x6a09e667U, 0xbb67ae85U, 0x3c6ef372U, 0xa54ff53aU,
		0x510e527fU, 0x9b05688cU, 0x1f83d9abU, 0x5be0cd19U
	};
	int fullblocks = datasize / 64;
	for(i = 0; i <= 15; i++)
	{
		w[i] = data[i + fullblocks * 16];
	}

	w[(datasize%64-4)/4] = nonce;
	for(i = 16; i <= 63; i++)
	{
		s0 = rrot(w[i - 15], 7) ^ rrot(w[i - 15], 18) ^ (w[i - 15] >> 3);
		s1 = rrot(w[i - 2], 17) ^ rrot(w[i - 2], 19) ^ (w[i - 2] >> 10);
		w[i] = w[i - 16] + s0 + w[i - 7] + s1;
	}
	a = midstate[0];
	b = midstate[1];
	c = midstate[2];
	d = midstate[3];
	e = midstate[4];
	f = midstate[5];
	g = midstate[6];
	h = midstate[7];
	for(i = 0; i <= 63; i++)
	{
		s0 = rrot(a, 2) ^ rrot(a, 13) ^ rrot(a, 22);
		maj = (a & b) ^ (a & c) ^ (b & c);
		t2 = s0 + maj;
		s1 = rrot(e, 6) ^ rrot(e, 11) ^ rrot(e, 25);
		ch = (e & f) ^ ((~e) & g);
		t1 = h + s1 + ch + k[i] + w[i];
		h = g;
		g = f;
		f = e;
		e = d + t1;
		d = c;
		c = b;
		b = a;
		a = t1 + t2;
	}
	w[0] = a + midstate[0];
	w[1] = b + midstate[1];
	w[2] = c + midstate[2];
	w[3] = d + midstate[3];
	w[4] = e + midstate[4];
	w[5] = f + midstate[5];
	w[6] = g + midstate[6];
	w[7] = h + midstate[7];
	w[8] = 0x80000000U;
	for(i = 9; i <= 14; i++)
		w[i] = 0U;
	w[15] = 0x0100U;
	for(i = 16; i <= 63; i++)
	{
		s0 = rrot(w[i - 15], 7) ^ rrot(w[i - 15], 18) ^ (w[i - 15] >> 3);
		s1 = rrot(w[i - 2], 17) ^ rrot(w[i - 2], 19) ^ (w[i - 2] >> 10);
		w[i] = w[i - 16] + s0 + w[i - 7] + s1;
	}
	a = hc[0];
	b = hc[1];
	c = hc[2];
	d = hc[3];
	e = hc[4];
	f = hc[5];
	g = hc[6];
	h = hc[7];
	for(i = 0; i <= 63; i++)
	{
		s0 = rrot(a, 2) ^ rrot(a, 13) ^ rrot(a, 22);
		maj = (a & b) ^ (a & c) ^ (b & c);
		t2 = s0 + maj;
		s1 = rrot(e, 6) ^ rrot(e, 11) ^ rrot(e, 25);
		ch = (e & f) ^ ((~e) & g);
		t1 = h + s1 + ch + k[i] + w[i];
		h = g;
		g = f;
		f = e;
		e = d + t1;
		d = c;
		c = b;
		b = a;
		a = t1 + t2;
	}
	output[0] = a + hc[0];
	output[1] = b + hc[1];
	output[2] = c + hc[2];
	output[3] = d + hc[3];
	output[4] = e + hc[4];
	output[5] = f + hc[5];
	output[6] = g + hc[6];
	output[7] = h + hc[7];
}

void pascal_midstate(const uint32_t *data, uint32_t *hc)
{
	int i;
	uint32_t s0, s1, t1, t2, maj, ch, a, b, c, d, e, f, g, h;
	uint32_t w[64];

	const uint32_t k[64] = {
		0x428a2f98U, 0x71374491U, 0xb5c0fbcfU, 0xe9b5dba5U, 0x3956c25bU, 0x59f111f1U, 0x923f82a4U, 0xab1c5ed5U,
		0xd807aa98U, 0x12835b01U, 0x243185beU, 0x550c7dc3U, 0x72be5d74U, 0x80deb1feU, 0x9bdc06a7U, 0xc19bf174U,
		0xe49b69c1U, 0xefbe4786U, 0x0fc19dc6U, 0x240ca1ccU, 0x2de92c6fU, 0x4a7484aaU, 0x5cb0a9dcU, 0x76f988daU,
		0x983e5152U, 0xa831c66dU, 0xb00327c8U, 0xbf597fc7U, 0xc6e00bf3U, 0xd5a79147U, 0x06ca6351U, 0x14292967U,
		0x27b70a85U, 0x2e1b2138U, 0x4d2c6dfcU, 0x53380d13U, 0x650a7354U, 0x766a0abbU, 0x81c2c92eU, 0x92722c85U,
		0xa2bfe8a1U, 0xa81a664bU, 0xc24b8b70U, 0xc76c51a3U, 0xd192e819U, 0xd6990624U, 0xf40e3585U, 0x106aa070U,
		0x19a4c116U, 0x1e376c08U, 0x2748774cU, 0x34b0bcb5U, 0x391c0cb3U, 0x4ed8aa4aU, 0x5b9cca4fU, 0x682e6ff3U,
		0x748f82eeU, 0x78a5636fU, 0x84c87814U, 0x8cc70208U, 0x90befffaU, 0xa4506cebU, 0xbef9a3f7U, 0xc67178f2U
	};

	for(i = 0; i <= 15; i++)
	{
		w[i] = data[i];
	}
	for(i = 16; i <= 63; i++)
	{
		s0 = ROTR32(w[i - 15], 7) ^ ROTR32(w[i - 15], 18) ^ (w[i - 15] >> 3);
		s1 = ROTR32(w[i - 2], 17) ^ ROTR32(w[i - 2], 19) ^ (w[i - 2] >> 10);
		w[i] = w[i - 16] + s0 + w[i - 7] + s1;
	}
	a = hc[0];
	b = hc[1];
	c = hc[2];
	d = hc[3];
	e = hc[4];
	f = hc[5];
	g = hc[6];
	h = hc[7];
	for(i = 0; i <= 63; i++)
	{
		s0 = ROTR32(a, 2) ^ ROTR32(a, 13) ^ ROTR32(a, 22);
		maj = (a & b) ^ (a & c) ^ (b & c);
		t2 = s0 + maj;
		s1 = ROTR32(e, 6) ^ ROTR32(e, 11) ^ ROTR32(e, 25);
		ch = (e & f) ^ ((~e) & g);
		t1 = h + s1 + ch + k[i] + w[i];
		h = g;
		g = f;
		f = e;
		e = d + t1;
		d = c;
		c = b;
		b = a;
		a = t1 + t2;
	}
	hc[0] += a;
	hc[1] += b;
	hc[2] += c;
	hc[3] += d;
	hc[4] += e;
	hc[5] += f;
	hc[6] += g;
	hc[7] += h;
}

int scanhash_pascal(int thr_id, uint32_t *pdata, uint32_t datasize,
					uint32_t *ptarget, uint32_t max_nonce,
					uint32_t *hashes_done)
{
	static THREAD uint32_t *result = nullptr;
	static THREAD volatile bool init = false;
	static THREAD uint32_t *pdata2 = nullptr;

	const uint32_t first_nonce = pdata[datasize / 4 - 1];
	uint32_t throughput = device_intensity(device_map[thr_id], __func__, 1U << 28);
	throughput = min(throughput, (max_nonce - first_nonce + 1));

	if(opt_benchmark)
		ptarget[7] = 0x0005;

	if(!init)
	{
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		hipDeviceSetCacheConfig(hipFuncCachePreferL1);

		pascal_cpu_init(thr_id);
		CUDA_SAFE_CALL(hipHostMalloc(&result, 2 * sizeof(uint32_t)));
		CUDA_SAFE_CALL(hipHostMalloc(&pdata2, datasize+64));
		result[0] = 0; result[1] = 0;
		init = true;
	}
	const int chunks = datasize / 64;
	const int datasize32 = datasize / 4;

	// change endianness
	for(size_t i = 0; i < datasize32-1; i ++)
		*(pdata2+i) = swab32(*(pdata+i));
	pdata2[datasize32 - 1] = pdata[datasize32 - 1];

	// padding overflow
	if(datasize % 64 > 53)
	{
		applog(LOG_ERR, "Error: data size %d is not being supported yet", datasize);
		sleep(1);
		return 0;
	}

	// add padding
	if(datasize % 64 > 0)
	{
		pdata2[datasize32] = 0x80000000;
		for(int i = 1; i <= 16 - datasize % 64 / 4; i++)
			pdata2[datasize32 + i] = 0;
		pdata2[chunks * 16 + 15] = datasize * 8;
	}

	uint32_t ms[8] =
	{
		0x6a09e667U, 0xbb67ae85U, 0x3c6ef372U, 0xa54ff53aU,
		0x510e527fU, 0x9b05688cU, 0x1f83d9abU, 0x5be0cd19U
	};

	if(chunks > 0)
		pascal_midstate(pdata2, ms);
	if(chunks > 1)
		pascal_midstate(pdata2 + 16, ms);
	if(chunks > 2)
		pascal_midstate(pdata2 + 32, ms);

	copydata(pdata2 + chunks * 16);
	do
	{
		pascal_cpu_hash(thr_id, throughput, pdata2[datasize32 - 1], (datasize % 64) - 4, ms, result);

		if(stop_mining)
		{
			mining_has_stopped[thr_id] = true;
			pthread_exit(nullptr);
		}
		if(result[0] != 0)
		{
			uint32_t vhash64[8] = {0};
			pascal_hash(vhash64, pdata2, datasize, result[0], ms);

			if(!opt_verify || (vhash64[0] == 0 && fulltest_pascal(vhash64, ptarget)))
			{
				int res = 1;
				// check if there was some other ones...
				*hashes_done = pdata2[datasize32 - 1] - first_nonce + throughput;
				if(result[1] != 0)
				{
					pascal_hash(vhash64, pdata2, datasize, result[1], ms);
					if(!opt_verify || (vhash64[0] == 0 && fulltest_pascal(vhash64, ptarget)))
					{
						pdata[datasize32 + 1] = result[1];
						res++;
						if(opt_benchmark)
							applog(LOG_INFO, "GPU #%d Found second nounce %08x", device_map[thr_id], result[1]);
					}
					else
					{
						if(vhash64[0] > 0)
						{
							applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], result[1]);
						}
					}
				}
				pdata[datasize32 - 1] = result[0];
				if(opt_benchmark)
					applog(LOG_INFO, "GPU #%d Found nounce %08x", device_map[thr_id], result[0]);
				return res;
			}
			else
			{
				if(vhash64[0] > 0)
				{
					applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], result[0]);
				}
			}
		}

		pdata2[datasize32 - 1] += throughput;
	} while(!work_restart[thr_id].restart && max_nonce - throughput > pdata2[datasize32 - 1]);

	*hashes_done = pdata2[datasize32 - 1] - first_nonce;

	return 0;
}
